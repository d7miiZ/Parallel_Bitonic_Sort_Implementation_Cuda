
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <inttypes.h>

__global__ void sort(unsigned long long *a, int step, int stage, unsigned long long sl, unsigned long long N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int shift = N / 2;
    int on = (index % N) < (N / 2);
    int ascinding = (index / sl) % 2 == 0 ? 1 : 0;

    if (on)
    {
        if (ascinding)
        {
            if (a[index] > a[index + shift])
            {
                unsigned long long temp = a[index];
                a[index] = a[index + shift];
                a[index + shift] = temp;
            }
        }
        else
        {
            if (a[index] < a[index + shift])
            {
                unsigned long long temp = a[index];
                a[index] = a[index + shift];
                a[index + shift] = temp;
            }
        }
    }
}

int main(void)
{
    unsigned long long *a;
    unsigned long long *d_a;
    int steps;
    int i, j;
    int dev;
    int threads, block;
    hipDeviceProp_t prop;

    hipGetDevice(&dev);
    hipGetDeviceProperties(&prop, dev);

    printf("Choose A Number For x in 2^x: ");
    scanf("%d", &steps);
    unsigned long long n = pow(2, steps);
    
    printf("\nNumber Of Elements Will Be %llu", n);
    unsigned long long size = n * sizeof(unsigned long long);

    if (n > prop.maxThreadsPerBlock)
    {
        threads = prop.maxThreadsPerBlock;
        block = n / prop.maxThreadsPerBlock;
    }
    else
    {
        threads = n;
        block = 1;
    }
    hipMalloc((void **)&d_a, size);

    a = (unsigned long long *)malloc(size);

    uint64_t num;
    for (i = 0; i < n; i++)
    {
        num = rand();
        
        a[i] = num;
    }

    printf("\nArray Before Sorting:\n");
    for (j = 0; j < n; ++j)
        printf("%llu\n", a[j]);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);


    int stage;
    int step;
    for (step = 1; step <= steps; step++)
    {
        unsigned long long sl = pow(2, step);
        for (stage = 1; stage <= step; stage++)
        {
            unsigned long long N = sl / (pow(2, stage - 1));
            sort<<<block, threads>>>(d_a, step, stage, sl, N);
        }
    }

    hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);

    printf("\nThe sorted array:\n");
    for (j = 0; j < n; ++j)
        printf("%llu\n", a[j]);

    free(a);
    hipFree(d_a);
    return 0;
}
